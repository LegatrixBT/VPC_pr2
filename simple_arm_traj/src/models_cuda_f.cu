#include "hip/hip_runtime.h"
// /*----------------------------------------------------------
// Adrien Durand-Petiteville
// Universidade Federal de Pernambuco
// June 2019
// ----------------------------------------------------------*/

#include "../include/simple_arm_traj/models_cuda_f.h"

////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////
void setupModelDataCuda_f(xyz_model_data_cuda_f* func_data_cuda, int nbPrediction, int nbFeatures, float ts, float dx, float focal, float* h_XYref)
{

  // Allocate host memory
  // func_data_cuda->v_X = new float[nbFeatures];
  // func_data_cuda->v_Y = new float[nbFeatures];
  // func_data_cuda->v_z = new float[nbFeatures];
  func_data_cuda->v_XYz = new float[3*nbFeatures];
  func_data_cuda->parameters = new float[5];
  func_data_cuda->h_command = new float[3*nbPrediction];
  func_data_cuda->h_Cgrad = new float[3*nbPrediction];
  func_data_cuda->h_commandDelta = new float[3*nbPrediction*3*nbPrediction];
  func_data_cuda->h_predictedState = new float[4*(nbPrediction+1)];

  func_data_cuda->h_G_finalConstraint = new double[3*nbPrediction];

  // Copy host data
  func_data_cuda->nbPrediction = nbPrediction;
  func_data_cuda->nbFeatures = nbFeatures;
  func_data_cuda->ts = ts;
  func_data_cuda->dx = dx;
  func_data_cuda->focal = focal;
  func_data_cuda->parameters[0] = nbPrediction;
  func_data_cuda->parameters[1] = nbFeatures;
  func_data_cuda->parameters[2] = ts;
  func_data_cuda->parameters[3] = dx;
  func_data_cuda->parameters[4] = focal;

  // Allocate device memory
  hipMalloc(&func_data_cuda->d_command, 3*nbPrediction*sizeof(float));
  hipMalloc(&func_data_cuda->d_command2, 3*nbPrediction*sizeof(float));
  hipMalloc(&func_data_cuda->d_param, 5*sizeof(float));
  hipMalloc(&func_data_cuda->d_XYz, 3*nbFeatures*sizeof(float));
  hipMalloc(&func_data_cuda->d_XYref, 2*nbFeatures*sizeof(float));
  hipMalloc(&func_data_cuda->d_thetaP, sizeof(float));

  // hipMalloc(&func_data_cuda->d_C, sizeof(float));
  hipMalloc(&func_data_cuda->d_C, nbPrediction*nbFeatures*sizeof(float));

  hipMalloc(&func_data_cuda->d_Cgrad, 3*nbPrediction*sizeof(float));
  hipMalloc(&func_data_cuda->d_commandDelta, 3*nbPrediction*3*nbPrediction*sizeof(float));
  hipMalloc(&func_data_cuda->d_commandEqDelta, 3*nbPrediction*3*nbPrediction*sizeof(float));

  hipMalloc(&func_data_cuda->d_deltaTTP, 2*(nbPrediction+1)*(nbPrediction+1)*sizeof(float));
  hipMalloc(&func_data_cuda->d_deltaXY, 6*nbPrediction*nbPrediction*sizeof(float));

  hipMalloc(&func_data_cuda->d_CMat, 3*nbPrediction*nbPrediction*nbFeatures*sizeof(float));

  // Copy device data
  hipMemcpy(func_data_cuda->d_param, func_data_cuda->parameters, 5*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(func_data_cuda->d_XYref, h_XYref, 2*nbFeatures*sizeof(float), hipMemcpyHostToDevice);

}

/////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////
double xyz_model_wrap_cuda_f(unsigned n, const double *x, double *grad, void *my_func_data)
{

    // Cost function
    double h_C = 0.0;

    // Get the data from the structure
    xyz_model_data_cuda_f *d = (xyz_model_data_cuda_f *) my_func_data;

    // Count the number of calls
    d->iteration = d->iteration + 1;

const clock_t begin_time_func = clock();

    // Cast the command vector
    float* xx = new float[3*d->nbPrediction];
    for(int idx = 0; idx < 3*d->nbPrediction;idx++){xx[idx] = (float)x[idx];}

// cout << xx[0] << " " << xx[1] << " " << xx[2] << endl;

    // Compute the equivalent speeds
    equivalentSpeed(d->nbPrediction, xx, d->h_command, d->theta, d->thetaP, d->dx, d->ts);

    // Copy the data from host to device
    hipMemcpy(d->d_command, d->h_command, 3*d->nbPrediction*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d->d_XYz, d->v_XYz, 3*d->nbFeatures*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d->d_thetaP,&d->thetaP, sizeof(float), hipMemcpyHostToDevice);

    // Parallel computing of the cost function
int threadDim_model = d->nbFeatures*d->nbPrediction; // TBDC for a large number of features
// const clock_t begin_time_func_cuda = clock();
    xyz_model_cuda<<<1,threadDim_model>>>(d->d_C, d->d_command, d->d_XYz, d->d_XYref, d->d_param, d->thetaP, d->nbFeatures);
float* h_CMat = new float[d->nbFeatures*d->nbPrediction];
hipMemcpy(h_CMat, d->d_C, d->nbFeatures*d->nbPrediction*sizeof(float), hipMemcpyDeviceToHost);
float C_f = 0.0;
for(int idx = 0; idx < d->nbFeatures*d->nbPrediction; idx++){C_f += h_CMat[idx];}
h_C = (double) C_f;

// if(isnan(h_C))
// {
    // cout << "C " << h_C << endl;
//     // for(int idx = 0; idx < d->nbFeatures*d->nbPrediction; idx++){cout << h_CMat[idx] << " ";}
//     // cout << endl;
//     for(int idx = 0; idx < 3*d->nbPrediction; idx++){cout << d->h_command[idx] << " ";}
//     cout << endl;
// }


// d->timeFunctioncuda += float( clock () - begin_time_func_cuda ) /  CLOCKS_PER_SEC*1000;

// hipDeviceSynchronize();
d->timeFunction += float( clock () - begin_time_func ) /  CLOCKS_PER_SEC*1000;

/////////
// Compute the final constraint
if(d->finalConstraint)
{
  C_f = 0.0;
  for(int idx = d->nbFeatures*(d->nbPrediction-1); idx < d->nbFeatures*d->nbPrediction; idx++)
  {
    C_f += h_CMat[idx];
  }
  d->C_finalConstraint = (double)C_f;
}
/////////

    // Compute the gradient
    if (grad)
    {
      // Count the number of calls
      d->iteration_grad = d->iteration_grad + 1;


const clock_t begin_time_grad = clock();


      // Delta to compute the gradient
      float delta = 0.001;


// const clock_t begin_time_eq = clock();

hipMemcpy(d->d_command2, xx, 3*d->nbPrediction*sizeof(float), hipMemcpyHostToDevice);


int offsetTTP = (d->nbPrediction+1)*(d->nbPrediction+1);
int nb0 = d->nbPrediction*(d->nbPrediction + 1);
int thread0 = 1024;
int block0 = (nb0 / thread0) + 1;

computeDeltaTheta<<<block0,thread0>>>(d->d_command2, d->d_deltaTTP, offsetTTP, d->nbPrediction, d->ts, delta, d->theta, d->thetaP);

sumDelta<<<1,2*(d->nbPrediction+1)>>>(d->d_deltaTTP, d->nbPrediction);

int offsetXY = 3*d->nbPrediction*d->nbPrediction;
int nb1 = 3*d->nbPrediction*d->nbPrediction;
int thread1 = 1024;
int block1 = (nb1 / thread1) + 1;

computeDeltaXY<<<block1,thread1>>>(d->d_deltaXY, d->d_command2, d->d_deltaTTP, offsetXY, d->nbPrediction, d->ts, d->dx, delta);

//**
//
// float* debug2 = new float[6*d->nbPrediction*d->nbPrediction];
// hipMemcpy(debug2, d->d_deltaXY, 6*(d->nbPrediction)*(d->nbPrediction)*sizeof(float), hipMemcpyDeviceToHost);
//
// for(int i = 0; i < 6*(d->nbPrediction)*(d->nbPrediction); i++)
// {
//   cout << debug2[i] << " ";
//   if((i+1)%(d->nbPrediction) == 0){cout << endl;}
//   if((i+1)%(d->nbPrediction*d->nbPrediction) == 0){cout << endl;}
// }
//**


sumDelta<<<1,6*d->nbPrediction>>>(d->d_deltaXY, d->nbPrediction-1);

eqSpeed_cuda<<<block1,thread1>>>(d->d_commandEqDelta, d->d_deltaXY, offsetXY, d->d_deltaTTP, offsetTTP, d->nbPrediction, d->ts, d->dx);

// hipDeviceSynchronize();//!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!1

// d->timeEq += float( clock () - begin_time_eq ) /  CLOCKS_PER_SEC*1000;

int nb2 = 3*d->nbFeatures*d->nbPrediction*d->nbPrediction;
int thread2 = 1024;
int block2 = (nb2 / thread2) + 1;

// hipDeviceSynchronize();//!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!1
// const clock_t begin_time_gradcuda = clock();
//       // xyz_gradient_cuda<<<gridDim_grad,threadDim_grad>>>(d->d_Cgrad, d->d_commandEqDelta, d->d_XYz, d->d_XYref, d->d_param, d->thetaP);
int thread3 = ceil((float)(3*d->nbPrediction)/32.0)*32;
// // cout << "asd "  << 3*d->nbPrediction << " " << thread3 << endl;
      zeros_cuda<<<1,thread3>>>(d->d_Cgrad, 3*d->nbPrediction);
//       // xyz_gradient_cuda<<<block2,thread2>>>(d->d_Cgrad, d->d_commandEqDelta, d->d_XYz, d->d_XYref, d->d_param, d->thetaP);
      xyz_gradient_cuda2<<<block2,thread2>>>(d->d_CMat, d->d_commandEqDelta, d->d_XYz, d->d_XYref, d->d_param, d->thetaP);

// float* pp = new float[3*d->nbFeatures*d->nbPrediction*d->nbPrediction];
// hipMemcpy(pp, d->d_CMat, 3*d->nbFeatures*d->nbPrediction*d->nbPrediction*sizeof(float), hipMemcpyDeviceToHost);


      sumC<<<1,3*d->nbPrediction>>>(d->d_Cgrad, d->d_CMat, d->nbPrediction, d->nbFeatures);
// hipDeviceSynchronize();//!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!1
// d->timeGradcuda += float( clock () - begin_time_gradcuda ) /  CLOCKS_PER_SEC*1000;

      // Retrieve the delta cost function
      hipMemcpy(d->h_Cgrad, d->d_Cgrad, 3*d->nbPrediction*sizeof(float), hipMemcpyDeviceToHost);

      // Compute the gradient
      for(int idx = 0; idx < 3*d->nbPrediction; idx++)
      {
        grad[idx] = (double)((d->h_Cgrad[idx] - h_C)/delta);

// if(isnan(grad[0]))
// {
// printf("%f %f  %f %f %f %f\n", grad[0], grad[1], grad[2], grad[3], grad[4], grad[5]);
// printf("A: %f %f %f %f %f\n", d->h_Cgrad[idx], h_C, delta, (d->h_Cgrad[idx] - h_C), (d->h_Cgrad[idx] - h_C)/delta);
// // printf("A: ");
// // for(int ddd = 0; ddd < 3*d->nbPrediction; ddd++)
// // {
// //   printf("%f ", d->h_Cgrad[ddd]);
// // }
// // printf("\n");
//
// }
      }

///////////////////////////////////
// Gradient of the final constraint
if(d->finalConstraint)
{
  zeros_cuda<<<1,thread3>>>(d->d_Cgrad, 3*d->nbPrediction);
  sum_finalconstraint<<<1,3*d->nbPrediction>>>(d->d_Cgrad, d->d_CMat, d->nbPrediction, d->nbFeatures);
  // Retrieve the delta cost function
  hipMemcpy(d->h_Cgrad, d->d_Cgrad, 3*d->nbPrediction*sizeof(float), hipMemcpyDeviceToHost);

  // Compute the gradient
  for(int idx = 0; idx < 3*d->nbPrediction; idx++)
  {
    d->h_G_finalConstraint[idx] = (double)((d->h_Cgrad[idx] - d->C_finalConstraint)/delta);
  }
}

d->timeGrad += float( clock () - begin_time_grad ) /  CLOCKS_PER_SEC*1000;
///////////////////////////////////
    }// If grad

  return h_C;

}// myfunc

////////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////////////
double xyz_constraint_cuda_f_wrap(unsigned n, const double *x, double *grad, void *my_func_data)
{
  // Cast
  xyz_model_data_cuda_f *d = (xyz_model_data_cuda_f *) my_func_data;

  // Retrieve the final constraint grad
  if (grad)
  {
    for(int idx = 0; idx < 3*d->nbPrediction; idx++)
    {
      grad[idx] = d->h_G_finalConstraint[idx];
    }
  }

  // Retrieve the final constraint cost
  return d->C_finalConstraint;
}

////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////
void equivalentSpeed(int nbPred, float* speedIn, float* speedOut, float thetaI, float thetaPI, float dx, float ts)
{
  // Camera position
  float xci = dx*cos(thetaI);
  float yci = dx*sin(thetaI);
  float xc = xci;
  float yc = yci;

  // Platform orientation
  float theta = thetaI;
  float thetaP = thetaPI;

  // Robot control inputs
  float v, w, wp;

  // Constants
  float c1, c2;
  float DX, DY;

  for(int idxPred = 0; idxPred < nbPred; idxPred++)
  {
    // Retrieve the current commands
    v = speedIn[3*idxPred];
    w = speedIn[3*idxPred + 1];
    wp = speedIn[3*idxPred + 2];

    // Update the constants
    c1 = w*ts/2;
    c2 = (2*theta+w*ts)/2;

    // Update the camera position
    if(abs(w) > 0.0001)
    {
      xc += 2*v/w*sin(c1)*cos(c2) - 2*dx*sin(c1)*sin(c2);
      yc += 2*v/w*sin(c1)*sin(c2) + 2*dx*sin(c1)*cos(c2);
    }
    else
    {
      // cout << "straight" << endl;
      xc += v*cos(theta)*ts - w*dx*sin(theta)*ts;
      yc += v*sin(theta)*ts + w*dx*cos(theta)*ts;
    }

    // Update the orientation
    theta += w*ts;
    thetaP += wp*ts;

    // Compute the equivalent speeds
    DX = xc - xci;
    DY = yc - yci;

    speedOut[3*idxPred + 1] = 2/ts*atan2(-DX*sin(thetaI) + DY*cos(thetaI), 2*dx + DX*cos(thetaI) + DY*sin(thetaI));
    if(abs(speedOut[3*idxPred + 1]) < 0.000001)
    {
      speedOut[3*idxPred] = sqrt(pow(DX,2) + pow(DY,2)) / ts;
    }
    else
    {
      float rslt = pow(speedOut[3*idxPred + 1],2)*((pow(DX,2) + pow(DY,2)) /
                                 (4*pow(sin(speedOut[3*idxPred + 1]*ts/2),2)) - dx*dx);
      if(rslt < 0.000001)
      {
        speedOut[3*idxPred] = 0.0;
      }
      else
      {
        speedOut[3*idxPred] = sqrt(rslt);
      }

    }
    speedOut[3*idxPred + 2] = (thetaP - thetaPI + theta - thetaI - speedOut[3*idxPred + 1]*ts)/ts;

  }// for Pred

}


////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////
__global__
void xyz_model_cuda(float* C, float* command, float* XYz, float* XYref, float* param, float thetaP, int nbFeat)
{

  // Thread index
  // + blockIdx.x*blockDim.x
  int idx = threadIdx.x;

  if (idx < param[0]*param[1])
  {

    // Command index
    int idxCom = floor((float)idx/(float)nbFeat);

    // Visual features index
    int idxVF = fmodf((float)idx,(float)nbFeat);

    // Retrieve parameters
    float ts = param[2];
    float dx = param[3];
    float f = param[4];

    float A, t0, t1, c1, c2;
    float Xp, Yp, zp;
    float cx = 0.095837; ///////// Not the correct values
    float cy = 0.022; ///////// See robotClass.cpp

    // Current comand vector
    float v = command[idxCom*3];
    float w = command[idxCom*3 + 1];
    float wp = command[idxCom*3 + 2];

    // Current visual features
    float Xi = XYz[3*idxVF];
    float Yi = XYz[3*idxVF + 1];
    float zi = XYz[3*idxVF + 2];

    // Reference visual features
    float Xref = XYref[2*idxVF];
    float Yref = XYref[2*idxVF + 1];



    // Case #1
    if ((abs(w+wp) > 0.000001) & (abs(w) > 0.000001))
    {
      // cout << "CASE 1" << endl;
      A = (w+wp)*ts;
      t0 = thetaP;
      t1 = thetaP + wp*ts;

      c1 = Yi*zi/f - dx*sin(t0) - v/w*cos(t0) + cy;
      c2 = zi + dx*cos(t0) - v/w*sin(t0) + cx;

      zp = c1*sin(A) + c2*cos(A) - dx*cos(t1) + v/w*sin(t1) - cx;
      Xp = zi*Xi/zp;
      Yp = f/zp*(c1*cos(A) - c2*sin(A) + dx*sin(t1) + v/w*cos(t1) - cy);
    }// Case #1

    // Case #2
    else if ( (abs(w)<0.000001) && (abs(wp) > 0.000001))
    {
      // cout << "CASE 2" << endl;
      A = wp*ts;
      t0 = thetaP;
      t1 = thetaP + wp*ts;

      c1 = Yi*zi/f - v/(2*wp)*cos(t0) + cy;
      c2 = zi - v/(2*wp)*sin(t0) + cx;

      zp = c1*sin(A) + c2*cos(A) - v*ts*cos(t1) + v/(2*wp)*sin(t1) - cx;
      Xp = zi*Xi/zp;
      Yp = f/zp*(c1*cos(A) - c2*sin(A) + v*ts*sin(t1) + v/(2*wp)*cos(t1) -cy);
    }// Case #2

    // Case #3
    else if ((abs(w + wp) < 0.000001) && (abs(w) > 0.000001))
    {
      // cout << "CASE 3" << endl;
      t0 = thetaP;
      t1 = thetaP + wp*ts;

      zp = -v/wp*(sin(t1) - sin(t0)) + w/wp*dx*(cos(t1) - cos(t0)) + zi;
      Xp = zi*Xi/zp;
      Yp = (-f*v*(cos(t1) - cos(t0)) - f*w*dx*(sin(t1) - sin(t0)) + wp*zi*Yi) / (wp*zp);
    }// Case #3

    // Case #4
    else if ((abs(w)<0.000001) && (abs(wp)<0.000001))
    {
      t0 = thetaP;
      t1 = thetaP;

      zp = -v*cos(t0)*ts + zi;
      Xp = zi*Xi/zp;
      Yp = (-f*v*sin(t0)*ts + Yi*zi) / (zp);

    }// Case #4

    // Update the cost function
    C[idx] = (Xp - Xref)*(Xp - Xref) + (Yp - Yref)*(Yp - Yref);

  }// if thread
}

////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////
__global__
void xyz_gradient_cuda2(float* C, float* command, float* XYz, float* XYref, float* param, float thetaP)
{

  /////
  // Retrieve parameters
  int nbPrediction = (int) param[0];
  int nbFeat = (int) param[1];
  float ts = param[2];
  float dx = param[3];
  float f = param[4];

  int tIdx = threadIdx.x + blockIdx.x*blockDim.x;

  if(tIdx < (3*nbPrediction*nbPrediction*nbFeat))
  {

    int idxGrad = floor((float)tIdx/(float)(nbFeat*nbPrediction));
    int rest = fmodf((float)tIdx, (float)(nbFeat*nbPrediction));
    int idxPred = floor((float)rest / (float)nbFeat);
    int idxVF = fmodf((float)rest, (float)nbFeat);

    // Define constants
    float A, t0, t1, c1, c2;
    float Xp, Yp, zp;
    float cx = 0.095837; ///////// Not the correct values
    float cy = 0.022; ///////// See robotClass.cpp

    // Current comand vector
    float v = command[idxGrad*3*nbPrediction + idxPred*3];
    float w = command[idxGrad*3*nbPrediction + idxPred*3 + 1];
    float wp = command[idxGrad*3*nbPrediction + idxPred*3 + 2];

    // Current visual features
    float Xi = XYz[3*idxVF];
    float Yi = XYz[3*idxVF + 1];
    float zi = XYz[3*idxVF + 2];

    // Reference visual features
    float Xref = XYref[2*idxVF];
    float Yref = XYref[2*idxVF + 1];
// if(tIdx == 0){
// printf("%f %f %f %f %f %f %f %f \n", ts, dx, f, Xi, Yi, zi, Xref, Yref);
// }
    // Case #1
    if ((abs(w+wp) > 0.000001) & (abs(w) > 0.000001))
    {
      A = (w+wp)*ts;
      t0 = thetaP;
      t1 = thetaP + wp*ts;

      c1 = Yi*zi/f - dx*sin(t0) - v/w*cos(t0) + cy;
      c2 = zi + dx*cos(t0) - v/w*sin(t0) + cx;

      zp = c1*sin(A) + c2*cos(A) - dx*cos(t1) + v/w*sin(t1) - cx;
      Xp = zi*Xi/zp;
      Yp = f/zp*(c1*cos(A) - c2*sin(A) + dx*sin(t1) + v/w*cos(t1)) -cy;
    }// Case #1

    // Case #2
    else if ( (abs(w)<0.000001) && (abs(wp) > 0.000001))
    {
      A = wp*ts;
      t0 = thetaP;
      t1 = thetaP + wp*ts;

      c1 = Yi*zi/f - v/(2*wp)*cos(t0) + cy;
      c2 = zi - v/(2*wp)*sin(t0) + cx;

      zp = c1*sin(A) + c2*cos(A) - v*ts*cos(t1) + v/(2*wp)*sin(t1) - cx;
      Xp = zi*Xi/zp;
      Yp = f/zp*(c1*cos(A) - c2*sin(A) + v*ts*sin(t1) + v/(2*wp)*cos(t1) -cy);
    }// Case #2

    // Case #3
    else if ((abs(w + wp) < 0.000001) && (abs(w) > 0.000001))
    {
      t0 = thetaP;
      t1 = thetaP + wp*ts;

      zp = -v/wp*(sin(t1) - sin(t0)) + w/wp*dx*(cos(t1) - cos(t0)) + zi;
      Xp = zi*Xi/zp;
      Yp = (-f*v*(cos(t1) - cos(t0)) - f*w*dx*(sin(t1) - sin(t0)) + wp*zi*Yi) / (wp*zp);
    }// Case #3

    // Case #4
    else if ((abs(w)<0.000001) && (abs(wp)<0.000001))
    {
      t0 = thetaP;
      t1 = thetaP;

      zp = -v*cos(t0)*ts + zi;
      Xp = zi*Xi/zp;
      Yp = (-f*v*sin(t0)*ts + Yi*zi) / (zp);
    }// Case #4

// printf("G %f %f %f %f\n", Xp, Yp, zp, (Xp - Xref)*(Xp - Xref) + (Yp - Yref)*(Yp - Yref));

    C[idxGrad*nbFeat*nbPrediction + rest] = (Xp - Xref)*(Xp - Xref) + (Yp - Yref)*(Yp - Yref);

  }//if thread
}


///////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////

__global__
void computeDeltaTheta(float* speed, float* deltaTTP, int offset, int nbPrediction, float ts, float delta, float thetaI, float thetaPI)
{

  int tIdx = threadIdx.x + blockIdx.x*blockDim.x;

  if (tIdx < (nbPrediction*(nbPrediction+1)))
  {

    int rowIdx = tIdx / nbPrediction;
    int colIdx = fmodf(tIdx, nbPrediction);

    if(rowIdx == colIdx+1)
    {
      // Fill the first element
      deltaTTP[0] = thetaI;
      deltaTTP[offset] = thetaPI;

      // Fill the first colum of the current row (initial value)
      deltaTTP[rowIdx*(nbPrediction + 1)] = thetaI;
      deltaTTP[rowIdx*(nbPrediction + 1) + offset] = thetaPI;

      // Fill the diagonal with noise (rowIdx + 1 is here to deal with the first line without noise)
      deltaTTP[tIdx + rowIdx + 1] = (speed[3*colIdx + 1] + delta)*ts;
      deltaTTP[tIdx + rowIdx + 1 + offset] = (speed[3*colIdx + 2] + delta)*ts;

    }
    else
    {
      // Compute delta theta
      deltaTTP[tIdx + rowIdx + 1] = speed[3*colIdx + 1]*ts;
      deltaTTP[tIdx + rowIdx + 1 + offset] = speed[3*colIdx + 2]*ts;
    }
  }
}

///////////////////////////////////////////
///////////////////////////////////////////

__global__
void sumDelta(float* delta, int nbPrediction)
{
  // Compute the offset to acces next row
  int offset = threadIdx.x*(nbPrediction + 1);

  // Sum each element of the row vector
  for(int idx = 0; idx < nbPrediction; idx++)
  {
    delta[offset + idx + 1] += delta[offset + idx];
  }

}

///////////////////////////////////////////////////////
///////////////////////////////////////////////////////
__global__
void sumC(float* C, float* Cmat, int nbPrediction, int nbFeatures)
{
  if(threadIdx.x < 3*nbPrediction)
  {

    // Compute the offset to access next row of the gradient
    int offset = threadIdx.x*nbPrediction*nbFeatures;

    // Sum each element of the row
    float sum = 0.0;
    for(int idx = 0; idx < nbPrediction*nbFeatures; idx++)
    {
      sum += Cmat[offset + idx];
    }

    C[threadIdx.x] = sum;
  }
}

///////////////////////////////////////////////////////
///////////////////////////////////////////////////////
__global__
void sum_finalconstraint(float* C, float* Cmat, int nbPrediction, int nbFeatures)
{
  if(threadIdx.x < 3*nbPrediction)
  {

    // Compute the offset to access next row of the gradient
    int offset = threadIdx.x*nbPrediction*nbFeatures;

    // Sum each element of the row
    float sum = 0.0;
    for(int idx = 0; idx < nbFeatures; idx++)
    {
      sum += Cmat[offset + idx*nbPrediction + nbPrediction - 1];
    }
    C[threadIdx.x] = sum;
  }
}

/////////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////////
__global__
void computeDeltaXY(float* deltaXY, float* speed, float* deltaTTP, int offset, int nbPrediction, float ts, float dx, float delta)
{

  // Thread index
  int tIdx = threadIdx.x + blockIdx.x*blockDim.x;

  if(tIdx < 3*nbPrediction*nbPrediction)
  {

    //
    int rowOutIdx = tIdx / nbPrediction;
    int rowInIdx = rowOutIdx / 3 + 1;
    int colIdx = fmodf(tIdx, nbPrediction);
    int code = fmodf(rowOutIdx,3);

    // Declare parameters
    float v, w, theta, c1, c2;

    // Select the relevant parameters
    if(code == 0)
    {
      if(rowInIdx == colIdx + 1)
      {
        v = speed[3*colIdx] + delta;
      }
      else
      {
        v = speed[3*colIdx];
      }

      w = speed[3*colIdx + 1];

      theta = deltaTTP[colIdx];

    }
    else if(code == 1)
    {
      v = speed[3*colIdx];

      if(rowInIdx == colIdx + 1)
      {
        w = speed[3*colIdx + 1] + delta;;
      }
      else
      {
        w = speed[3*colIdx + 1];
      }

      theta = deltaTTP[rowInIdx*(nbPrediction + 1) + colIdx];

    }
    else
    {
      v = speed[3*colIdx];
      w = speed[3*colIdx + 1];
      theta =  deltaTTP[colIdx];
    }

    // Compute DeltaX and DeltaY
    c1 = w*ts/2;
    c2 = (2*theta+w*ts)/2;

    if(abs(w) > 0.000001)
    {
      deltaXY[tIdx] = 2*v/w*sin(c1)*cos(c2) - 2*dx*sin(c1)*sin(c2);
      deltaXY[tIdx + offset] = 2*v/w*sin(c1)*sin(c2) + 2*dx*sin(c1)*cos(c2);
    }
    else
    {
      // cout << "straight" << endl;
      deltaXY[tIdx] = v*cos(theta)*ts - w*dx*sin(theta)*ts;
      deltaXY[tIdx + offset] = v*sin(theta)*ts + w*dx*cos(theta)*ts;
    }
  }
}


///////////////////////////////////////////////////////////
__global__
void eqSpeed_cuda(float* speedOut, float* deltaXY, int offsetXY, float* deltaTTP, int offsetTTP, int nbPrediction, float dx, float ts)
{

  // Thread index -> match the output format
  int tIdx = threadIdx.x + blockIdx.x*blockDim.x;

  if(tIdx < 3*nbPrediction*nbPrediction)
  {

    int row = tIdx / nbPrediction;
    int col = fmodf(tIdx, nbPrediction);
    int rowTTP = row / 3 + 1;
    int mode  = fmodf(row, 3);

    float DX = deltaXY[tIdx];
    float DY = deltaXY[tIdx + offsetXY];
    float thetaI = deltaTTP[0];
    float thetaPI = deltaTTP[offsetTTP];
    float theta, thetaP;


    if(mode == 0) // No noise
    {
      theta = deltaTTP[col + 1];
      thetaP = deltaTTP[col + 1 + offsetTTP];
    }
    else if(mode == 1) // Noise on theta
    {
      theta = deltaTTP[rowTTP*(nbPrediction + 1) + col + 1];
      thetaP = deltaTTP[col + 1 + offsetTTP];
    }
    else // Noise on theta platine
    {
      theta = deltaTTP[col + 1];
      thetaP = deltaTTP[rowTTP*(nbPrediction + 1) + col + 1 + offsetTTP];
    }

    // w eq
    float w = 2/ts*atan2(-DX*sin(thetaI) + DY*cos(thetaI), 2*dx + DX*cos(thetaI) + DY*sin(thetaI));

    speedOut[3*tIdx + 1] = w;

    // v eq
    float v, rslt;
    if(abs(w) < 0.000001)
    {
      v = sqrt(pow(DX,2) + pow(DY,2)) / ts;
    }
    else
    {
      rslt = pow(w,2)*((pow(DX,2) + pow(DY,2)) / (4*pow(sin(w*ts/2),2)) - dx*dx);
      if(rslt < 0.000001)
      {
        v = 0.0;
      }
      else
      {
        v = sqrt(rslt);
      }


    }
    speedOut[3*tIdx + 0] = v;

    // wp eq
    float wp = (thetaP - thetaPI + theta - thetaI - w*ts)/ts;
    speedOut[3*tIdx + 2] = wp;
if(isnan(v)){printf(" EQ: %f | %f | %f %f %f %f %f %f \n", v, rslt, w, wp, DX, DX, theta, thetaP);}

  }

}



////////////////////////////////////////////
//
__global__
void zeros_cuda(float* vec, int size)
{
  if(threadIdx.x < size)
  {
    vec[threadIdx.x] = 0.0;
  }
}
